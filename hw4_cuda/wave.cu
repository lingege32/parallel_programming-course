/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <iostream>
#include <iomanip>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define THREADPERWARP 32
#define SMNUM		   80

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
	tpoints, 	     		/* total points along string */
	rcode;                  	/* generic return code */
float  values[MAXPOINTS]; 	/* values at time t */
float *cudaValues;
int cudaArraySize;


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
	char tchar[20];

	/* check number of points, number of iterations */
	while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
		printf("Enter number of points along vibrating string [%d-%d]: "
				,MINPOINTS, MAXPOINTS);
		scanf("%s", tchar);
		tpoints = atoi(tchar);
		if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
			printf("Invalid. Please enter value between %d and %d\n",
					MINPOINTS, MAXPOINTS);
	}
	while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
		printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
		scanf("%s", tchar);
		nsteps = atoi(tchar);
		if ((nsteps < 1) || (nsteps > MAXSTEPS))
			printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
	}

	printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
	int i, j;
	float x, tmp;

	/* Calculate initial values based on sine curve */
	//float fac = 2.0 * PI;
	float fac = 6.2831853;
	//k = 0.0;
	tmp = tpoints - 1;
	for (j = 0; j < tpoints; ++j) {
		x = static_cast<float>(j)/tmp;
		values[j] = sin (fac * x);
	}

	/* Initialize old values array */
}
/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
	int i;

	printf("0.0000 ");
	for (i = 1; i < tpoints; ++i) {
		printf("%6.4f ", values[i]);
		if (i%10 == 9)
			printf("\n");
	}
}
__global__ void cudaExecute(float* cudaAns,int howMany,int tpoints,int tIteration)
{
	float cudaValues,cudaOld,cudaNew;
	double cudaTwiceValue;
	double valuePar = 2.0 - static_cast<float>(0.09) * 2.0;
	for(int block=0;block<howMany;++block)
	{
		int ansIndex=block*(SMNUM*THREADPERWARP) + blockIdx.x*blockDim.x + threadIdx.x;
		cudaValues = cudaAns[ansIndex];
		cudaOld = cudaValues;
		for(int iter=0;iter<tIteration;++iter)
		{
			cudaTwiceValue = valuePar * cudaValues;
			cudaNew = (cudaTwiceValue) - cudaOld;
			//cudaNew = (2.0 * cudaValues)
			//				 - cudaOld
			//				 + (static_cast<float>(0.09) *  (-2.0)*cudaValues);
			cudaOld=cudaValues;
			cudaValues=cudaNew;
		}
		cudaAns[ansIndex]=cudaValues;
	}
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	int howManyBlock=tpoints/(SMNUM*THREADPERWARP);
	if(tpoints%(SMNUM*THREADPERWARP)!=0)
		++howManyBlock;
	cudaArraySize = tpoints*sizeof(float);
	printf("Initializing points on the line...\n");
	hipMalloc((void**)&cudaValues,cudaArraySize);
	init_line();
	hipMemcpy(cudaValues,values,cudaArraySize,hipMemcpyHostToDevice);
	cudaExecute<<<SMNUM,THREADPERWARP>>>(cudaValues,howManyBlock,tpoints,nsteps);
	hipMemcpy(values,cudaValues,cudaArraySize-4,hipMemcpyDeviceToHost);
	//update();
	printf("Updating all points for all time steps...\n");
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");

	return 0;
}
